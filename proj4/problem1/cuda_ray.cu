
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define SPHERES 20

#define rnd(x) (x * rand() / RAND_MAX)
#define INF 2e10f
#define DIM 2048

struct Sphere
{
    float r, b, g;
    float radius;
    float x, y, z;
    __device__ float hit(float ox, float oy, float *n)
    {
        float dx = ox - x;
        float dy = oy - y;
        if (dx * dx + dy * dy < radius * radius)
        {
            float dz = sqrtf(radius * radius - dx * dx - dy * dy);
            *n = dz / sqrtf(radius * radius);
            return dz + z;
        }
        return -INF;
    }
};

__global__ void kernel(Sphere *s, unsigned char *ptr)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int offset = x + y * DIM;
    float ox = (x - DIM / 2);
    float oy = (y - DIM / 2);

    float r = 0, g = 0, b = 0;
    float maxz = -INF;
    for (int i = 0; i < SPHERES; i++)
    {
        float n;
        float t = s[i].hit(ox, oy, &n);
        if (t > maxz)
        {
            float fscale = n;
            r = s[i].r * fscale;
            g = s[i].g * fscale;
            b = s[i].b * fscale;
            maxz = t;
        }
    }

    ptr[offset * 4 + 0] = (int)(r * 255);
    ptr[offset * 4 + 1] = (int)(g * 255);
    ptr[offset * 4 + 2] = (int)(b * 255);
    ptr[offset * 4 + 3] = 255;
}

void ppm_write(unsigned char *bitmap, int xdim, int ydim, FILE *fp)
{
    int i, x, y;
    fprintf(fp, "P3\n");
    fprintf(fp, "%d %d\n", xdim, ydim);
    fprintf(fp, "255\n");
    for (y = 0; y < ydim; y++)
    {
        for (x = 0; x < xdim; x++)
        {
            i = x + y * xdim;
            fprintf(fp, "%d %d %d ", bitmap[4 * i], bitmap[4 * i + 1], bitmap[4 * i + 2]);
        }
        fprintf(fp, "\n");
    }
}

int main(int argc, char *argv[])
{
    unsigned char *bitmap;
    const char *filename = "result.ppm";
    FILE *fp = fopen(filename, "w");
    Sphere *temp_s = (Sphere *)malloc(sizeof(Sphere) * SPHERES);
    Sphere *d_s;
    unsigned char *device_bitmap;
    hipEvent_t start;
    hipEvent_t end;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&end);
    srand(time(NULL));
    for (int i = 0; i < SPHERES; i++)
    {
        temp_s[i].r = rnd(1.0f);
        temp_s[i].g = rnd(1.0f);
        temp_s[i].b = rnd(1.0f);
        temp_s[i].x = rnd(2000.0f) - 1000;
        temp_s[i].y = rnd(2000.0f) - 1000;
        temp_s[i].z = rnd(2000.0f) - 1000;
        temp_s[i].radius = rnd(200.0f) + 40;
    }
    hipMalloc((void **)&d_s, sizeof(Sphere) * SPHERES);
    hipMalloc((void **)&device_bitmap, sizeof(unsigned char) * DIM * DIM * 4);
    hipMemcpy(d_s, temp_s, sizeof(Sphere) * SPHERES, hipMemcpyHostToDevice);
    hipEventRecord(start);
    kernel<<<dim3((DIM + 15) / 16, (DIM + 15) / 16), dim3(16, 16)>>>(d_s, device_bitmap);
    hipEventRecord(end);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsedTime, start, end);
    bitmap = (unsigned char *)malloc(sizeof(unsigned char) * DIM * DIM * 4);
    hipMemcpy(bitmap, device_bitmap, sizeof(unsigned char) * DIM * DIM * 4, hipMemcpyDeviceToHost);
    printf("CUDA ray tracing: %.3lf sec\n", elapsedTime);
    ppm_write(bitmap, DIM, DIM, fp);
    printf("[%s] was generated.", filename);
    fclose(fp);
    free(bitmap);
    free(temp_s);
    hipFree(d_s);
    hipEventDestroy(start);
    hipEventDestroy(end);
    hipFree(device_bitmap);
    return 0;
}
